
#include <hip/hip_runtime.h>
extern "C"



__global__ void add(double *deltas,double *in, double *out)
{
    uint32_t iStep = blockDim.x * gridDim.x ;
    // Assume squares here
//     uint32_t jStep = blockDim.y * gridDim.y ;
    uint32_t i = blockIdx.x * blockDim.x  + threadIdx.x;

    uint32_t j = (blockIdx.y * blockDim.y  + threadIdx.y)*iStep;

out[i+j] = in[i+j] + deltas[0]*deltas[1] *( -4 * in[i+j] + in[((blockIdx.x * blockDim.x  + threadIdx.x) -1)%iStep+j] + in[((blockIdx.x * blockDim.x  + threadIdx.x) +1)%iStep +j] + in[i+((blockIdx.y * blockDim.y  + threadIdx.y) -1)%iStep *iStep] + in[i+((blockIdx.y * blockDim.y  + threadIdx.y) +1)%iStep *iStep]) ;

}
