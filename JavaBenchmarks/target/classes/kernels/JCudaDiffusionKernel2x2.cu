
#include <hip/hip_runtime.h>
extern "C"



__global__ void add(double *deltas,double *in, double *out)
{
    uint32_t iStep = blockDim.x * gridDim.x*2 ;
    // Assume squares here
//     uint32_t jStep = blockDim.y * gridDim.y ;
       uint32_t i = (blockIdx.x * blockDim.x  + threadIdx.x)*2;
//
       uint32_t __i = (i-2)%iStep;
       uint32_t _i = (i-1)%iStep;
       uint32_t i_ = (i+1)%iStep;
       uint32_t j = (blockIdx.y * blockDim.y  + threadIdx.y)*2;
       uint32_t __j = (j -2)%iStep*iStep;
       uint32_t _j = (j- 1)%iStep*iStep;
       uint32_t j_ = (j + 1)%iStep*iStep;
       j*= iStep;
//        out[_i+_j] = in[ _i+_j];
//         out[_i+j] = in[ _i+j];
//          out[i+_j] = in[ i+_j];
//           out[i+j] = in[i+j];
out[_i+_j] = in[_i+_j] + .2 *( -4 * in[_i+_j] + in[__i+_j] + in[_i+__j] + in[i+_j] + in[_i+j]);
out[i+_j] =  in[i+_j] + .2 *( -4 * in[i+_j] + in[_i+_j] + in[i+__j] + in[i_+_j] + in[i+j]);
out[_i+j] =   in[_i+j] + .2 *( -4 * in[_i+j] +  in[__i+j] + in[_i+_j] + in[i+j] + in[_i+j_]);
out[i+j] = in[i+j] + .2*( -4 * in[i+j] + in[_i+j] + in[i+_j] + in[i_+j] + in[i+j_]);
}
