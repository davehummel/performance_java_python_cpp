
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

        sum[i,j] = j;

}
