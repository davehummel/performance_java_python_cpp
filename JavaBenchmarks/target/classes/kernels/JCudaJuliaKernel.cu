#include "hip/hip_runtime.h"
extern "C"
#include <thrust/complex.h>


__global__ void julia(double *complexAdditive,double *complexInputGrid, int *output)
{

    uint32_t i = blockIdx.x * blockDim.x  + threadIdx.x;


    uint16_t maxIterations = complexAdditive[0];
    double cReal = complexAdditive[1];
    double cImag = complexAdditive[2];

    double realPartSqr;
    double imagPartSqr;

    double z0 = complexInputGrid[i*2];
    double z1 = complexInputGrid[i*2+1];

    uint16_t n = 0;

    while (n < maxIterations && ((realPartSqr = z0 * z0) + (imagPartSqr = z1 * z1) < 4)) {
                z1 = 2 * z0 * z1 + cImag;
                z0 = realPartSqr - imagPartSqr + cReal;
                n++;
            }

    output[i] = n;

}
